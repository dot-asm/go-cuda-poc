
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel()
{
    printf("hello from GPU\n");
}

extern "C" __attribute__((visibility("default")))
void cuda_call()
{
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
}

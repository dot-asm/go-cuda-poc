
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel()
{
    printf("hello from GPU\n");
}

extern "C"
#ifdef _WIN32
__declspec(dllexport)
#else
__attribute__((visibility("default")))
#endif
void cuda_call()
{
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
}

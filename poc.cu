
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

__global__ void kernel()
{
    printf("hello from GPU\n");
}

struct Error {
    int code;
    char *message;
};

extern "C"
#ifdef _WIN32
__declspec(dllexport)
#else
__attribute__((visibility("default")))
#endif
Error cuda_call()
{
    kernel<<<1,1>>>();
    auto err = hipGetLastError();
    if (err != hipSuccess)
        return {err, strdup(hipGetErrorString(err))};
    err = hipDeviceSynchronize();
    return {err, strdup(hipGetErrorString(err))};
}
